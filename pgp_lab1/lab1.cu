
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define CSC(call)                                                                                             \
    do                                                                                                        \
    {                                                                                                         \
        hipError_t status = call;                                                                            \
        if (status != hipSuccess)                                                                            \
        {                                                                                                     \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status)); \
            exit(0);                                                                                          \
        }                                                                                                     \
    } while (0)

// идентификатор '__global__' говрит, что функция будет работать на gpu
__global__ void parallel_reverse(double *vec, double *vec_reverse, int n)
{
    // вычисляем абсолютный номер потока
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // вычисляем число потоков - это будет наш шаг, если потоков меньше чем n
    int offset = blockDim.x * gridDim.x;

    while (idx < n)
    {
        assert(idx < n);
        vec_reverse[idx] = vec[n - 1 - idx];
        idx += offset;
    }
}

int main()
{
    int n;
    scanf("%d", &n);

    double *vec = (double *)malloc(sizeof(double) * n);
    for (int i = 0; i < n; i++)
    {
        scanf("%lf", &vec[i]);
    }

    double *dev_vec;
    double *dev_vec_reverse;
    // выделяем память на gpu
    CSC(hipMalloc(&dev_vec, sizeof(double) * n));
    CSC(hipMalloc(&dev_vec_reverse, sizeof(double) * n));
    CSC(hipMemcpy(dev_vec, vec, sizeof(double) * n, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start));

    // <<<кол-во блоков, размер одного блока>>>
    // max -- <<<65535, 1024>>>
    parallel_reverse<<<256, 256>>>(dev_vec, dev_vec_reverse, n);
    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

    CSC(hipEventRecord(stop));
    CSC(hipEventSynchronize(stop));

    float t;
    CSC(hipEventElapsedTime(&t, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));

    printf("time = %f ms\n", t);

    CSC(hipMemcpy(vec, dev_vec_reverse, sizeof(double) * n, hipMemcpyDeviceToHost));

    // for (int i = 0; i < n; i++)
    // {
    //     printf("%.10e ", vec[i]);
    // }
    // printf("\n");

    CSC(hipFree(dev_vec_reverse));
    CSC(hipFree(dev_vec));

    free(vec);
    return 0;
}
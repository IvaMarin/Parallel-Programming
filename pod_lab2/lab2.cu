#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)                                                 \
	do                                                            \
	{                                                             \
		hipError_t res = call;                                   \
		if (res != hipSuccess)                                   \
		{                                                         \
			fprintf(stderr, "ERROR in %s:%d. Message: %s\n",      \
					__FILE__, __LINE__, hipGetErrorString(res)); \
			exit(0);                                              \
		}                                                         \
	} while (0)

struct comparator
{
	// переопределение оператора "()" для экземпляра этой структуры
	__host__ __device__ bool operator()(double a, double b)
	{
		return fabs(a) < fabs(b);
	}
};

// функция меняющая местами две строки
__global__ void swap_rows(double *system, int curr_id, int max_id, int n)
{
	// вычисляем абсолютный номер потока
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// вычисляем число потоков - это будет наш шаг, если потоков меньше чем n
	int offset = blockDim.x * gridDim.x;

	double element;
	for (int j = idx; j < n + 1; j += offset)
	{
		element = system[j * n + curr_id];
		system[j * n + curr_id] = system[j * n + max_id];
		system[j * n + max_id] = element;
	}
}

// функция "зануления" всех элементов ниже данного
__global__ void subtract_row_from_rows_below(double *system, int curr_id, int n)
{
	// вычисляем абсолютный номер потока
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	// вычисляем число потоков - это будет наш шаг, если потоков меньше чем n
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	double coefficient;
	for (int i = idx + curr_id + 1; i < n; i += offsetx)
	{
		coefficient = system[curr_id * n + i] / system[curr_id * n + curr_id];
		for (int j = idy + curr_id + 1; j < n + 1; j += offsety)
		{
			system[j * n + i] -= system[j * n + curr_id] * coefficient;
		}
	}
}

int main()
{
	int n; // размерность квадратной матрицы
	scanf("%d", &n);

	double *system = (double *)malloc(sizeof(double) * (n + 1) * n);
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			scanf("%lf", &system[i + j * n]); // сохраняем матрицу по столбцам
		}
	}
	for (int j = 0; j < n; j++)
	{
		scanf("%lf", &system[n * n + j]); // дописываем вектор свободных коэффициентов
	}

	double *dev_system;
	CSC(hipMalloc(&dev_system, sizeof(double) * (n + 1) * n));
	CSC(hipMemcpy(dev_system, system, sizeof(double) * (n + 1) * n, hipMemcpyHostToDevice));

	dim3 threadsperBlock(256);
	dim3 numBlocks(256);

	dim3 threadsperBlock2D(32, 32);
	dim3 numBlocks2D(256, 256);

	comparator compare_by_absolute_value;
	thrust::device_ptr<double> max_id_ptr;
	int max_id;

	hipEvent_t start, stop;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start));

	// прямой ход метода Гаусса
	for (int i = 0; i < n - 1; i++)
	{
		// выполняем приведение типов
		thrust::device_ptr<double> system_ptr(dev_system + i * n);

		// ищем максимум в массиве на GPU
		max_id_ptr = thrust::max_element(system_ptr + i, system_ptr + n, compare_by_absolute_value);
		max_id = max_id_ptr - system_ptr;

		if (i != max_id)
		{
			swap_rows<<<numBlocks, threadsperBlock>>>(dev_system, i, max_id, n);
			CSC(hipGetLastError());
		}
		subtract_row_from_rows_below<<<numBlocks2D, threadsperBlock2D>>>(dev_system, i, n);
		CSC(hipGetLastError());
	}

	CSC(hipEventRecord(stop));
	CSC(hipEventSynchronize(stop));

	float t1;
	CSC(hipEventElapsedTime(&t1, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
	
	CSC(hipMemcpy(system, dev_system, sizeof(double) * (n + 1) * n, hipMemcpyDeviceToHost));

	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start));

	// находим вектор неизветсных х
	double *x = (double *)malloc(sizeof(double) * n);
	for (int i = n - 1; i >= 0; i--)
	{
		x[i] = system[n * n + i];
		for (int j = n - 1; j > i; j--)
		{
			x[i] -= system[i + j * n] * x[j];
		}
		x[i] /= system[i * n + i];
	}

	CSC(hipEventRecord(stop));
	CSC(hipEventSynchronize(stop));

	float t2;
	CSC(hipEventElapsedTime(&t2, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));

	float t = t1 + t2;
	printf("time = %f ms\n", t);

	// for (int i = 0; i < n; i++)
	// {
	// 	printf("%.10e ", x[i]);
	// }
	// printf("\n");

	// Освобождаем память GPU
	CSC(hipFree(dev_system));

	// Освобождаем память СPU
	free(x);
	free(system);
	return 0;
}